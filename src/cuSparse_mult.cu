#include <stdio.h>
#include <iostream>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <cusp/print.h>
#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include "cuSparse_mult.cuh"

// error check macros
#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

inline void CudaSparseCheckCore(hipsparseStatus_t code, const char *file, int line) {
   if (code != HIPSPARSE_STATUS_SUCCESS) {
      fprintf(stderr,"Cuda Error %d : %s %s %d\n", code, hipsparseGetErrorString(code), file, line);
      exit(code);
   }
}

#define CudaSparseCheck( test ) { CudaSparseCheckCore((test), __FILE__, __LINE__); }

// perform sparse-matrix multiplication C=AxB
void cusparse_multiply(cusp::csr_matrix<int, float,cusp::device_memory> &A,
		cusp::csr_matrix<int, float,cusp::device_memory> &B, cusp::csr_matrix<int, float,cusp::device_memory> &res) {

	/* Get internal array pointers */
	int *A_row_offsets = thrust::raw_pointer_cast(A.row_offsets.data());
	int *B_row_offsets = thrust::raw_pointer_cast(B.row_offsets.data());
	int *A_column_indices = thrust::raw_pointer_cast(A.column_indices.data());
	int *B_column_indices = thrust::raw_pointer_cast(B.column_indices.data());
	float *A_values = thrust::raw_pointer_cast(A.values.data());
	float *B_values = thrust::raw_pointer_cast(B.values.data());

	/* Declare arrays for output matrix */
	int *C_row_offsets, *C_column_indices;
	float *C_values;


	/* Creates handles for matrices */
	hipsparseMatDescr_t descrA, descrB, descrC;
	hipsparseStatus_t stat;
	stat = hipsparseCreateMatDescr(&descrA);
	CUSPARSE_CHECK(stat);
	stat = hipsparseCreateMatDescr(&descrB);
	CUSPARSE_CHECK(stat);
	stat = hipsparseCreateMatDescr(&descrC);
	CUSPARSE_CHECK(stat);
	stat = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	CUSPARSE_CHECK(stat);
	stat = hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
	CUSPARSE_CHECK(stat);
	stat = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
	CUSPARSE_CHECK(stat);
	stat = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	CUSPARSE_CHECK(stat);
	stat = hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
	CUSPARSE_CHECK(stat);
	stat = hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);

	/* NO COMPILA, PERO COMPILABA CON LA CLASE ANTERIOR, LAS LLAMADAS A CUSPARSE ESTAN BIEN */
	int m = A.num_rows;
	int n = B.num_cols;
	int k = B.num_rows;
	hipsparseHandle_t handle;
	CudaSparseCheck(hipsparseCreate(&handle));

	// assume matrices A, B and D are ready.
	int baseC, nnzC;
	csrgemm2Info_t info = nullptr;
	size_t bufferSize;
	void *buffer = nullptr;
	// nnzTotalDevHostPtr points to host memory
	int *nnzTotalDevHostPtr = &nnzC;
	float alpha = 1.0;
	CudaSparseCheck(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	// step 1: create an opaque structure
	CudaSparseCheck(hipsparseCreateCsrgemm2Info(&info));

	// step 2: allocate buffer for csrgemm2Nnz and csrgemm2
	hipsparseMatDescr_t descr_D; // not used, created only for the sake of the arguments
	CudaSparseCheck(hipsparseCreateMatDescr(&descr_D));
	hipsparseScsrgemm2_bufferSizeExt(handle, m, n, k, &alpha, descrA, A.num_entries,
			A_row_offsets, A_column_indices, descrB, B.num_entries, B_row_offsets, B_column_indices, nullptr,
			descr_D, 0, nullptr, nullptr, info, &bufferSize);
	hipMalloc(&buffer, bufferSize);

	// step 3: compute C.row_ptr
	hipMalloc((void**) &C_row_offsets, sizeof(int) * (m + 1));
	hipsparseXcsrgemm2Nnz(handle, m, n, k, descrA, A.num_entries, A_row_offsets, A_column_indices,
			descrB, B.num_entries, B_row_offsets, B_column_indices,
			descr_D, 0, nullptr, nullptr, descrC, C_row_offsets, nnzTotalDevHostPtr,
			info, buffer);
	if (nullptr != nnzTotalDevHostPtr) {
		nnzC = *nnzTotalDevHostPtr;
	} else {
		hipMemcpy(&nnzC, C_row_offsets + m, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseC, C_row_offsets, sizeof(int), hipMemcpyDeviceToHost);
		nnzC -= baseC;
	}

	// step 4: finish sparsity pattern and value of C
	hipMalloc((void**) &C_column_indices, sizeof(int) * nnzC);
	hipMalloc((void**) &C_values, sizeof(float) * nnzC);
	// Remark: set C.val to nullptr if only sparsity pattern is required.
	hipsparseScsrgemm2(handle, m, n, k,
			&alpha, descrA, A.num_entries, A_values, A_row_offsets,
			A_column_indices, descrB, B.num_entries, B_values, B_row_offsets, B_column_indices,
			nullptr, descr_D, 0, nullptr, nullptr, nullptr,
			descrC, C_values, C_row_offsets, C_column_indices,
			info, buffer);
	hipDeviceSynchronize();

/* Solo para comparar */
	res.num_rows = m;
	res.num_cols = n;
	res.num_entries = nnzC;
	res.column_indices = cusp::array1d<int, cusp::device_memory>(C_column_indices, C_column_indices + nnzC);
	res.row_offsets = cusp::array1d<int, cusp::device_memory>(C_row_offsets, C_row_offsets + m + 1);
	res.values = cusp::array1d<float, cusp::device_memory>(C_values, C_values + nnzC);

	// step 5: destroy the opaque structure
	hipsparseDestroyCsrgemm2Info(info);
}
